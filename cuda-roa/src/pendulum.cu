#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>

#define NUM_SIMULATIONS 1000000
#define NUM_TIMESTEPS 1000
#define DT 0.01

// Pendulum parameters
#define LENGTH 1.0
#define MASS 1.0
#define GRAVITY 9.81

// PID parameters
#define KP 10.0
#define KI 0.0
#define KD 1.5
#define CONTROL_LIMIT 2.4

__global__ void init(unsigned int seed, hiprandState_t* states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= NUM_SIMULATIONS) return;

    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void simulate(hiprandState_t* states, double *state, double *initial_state, double *final_state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= NUM_SIMULATIONS) return;

    // Initial state
    double theta = hiprand_uniform_double(&states[idx]) * 2 * M_PI - M_PI; // -pi to pi
    double omega = hiprand_uniform_double(&states[idx]) * 4 - 2; // -1 to 1

    initial_state[idx*2] = theta;
    initial_state[idx*2+1] = omega;

    double integral = 0;
    double previous_error = 0;

    for (int t = 0; t < NUM_TIMESTEPS; t++) {
        // PID controller
        double error = 0 - theta; // error is difference from upright position
        integral += error * DT;
        double derivative = (error - previous_error) / DT;
        double control = KP * error + KI * integral + KD * derivative;
        previous_error = error;

        // Clip control to reasonable values
        if (control > CONTROL_LIMIT) control = CONTROL_LIMIT;
        if (control < -CONTROL_LIMIT) control = -CONTROL_LIMIT;

        // Pendulum dynamics
        double alpha = GRAVITY/LENGTH * sin(theta) + control/MASS/LENGTH/LENGTH;

        // Update state using Euler integration
        theta += DT * omega;
        omega += DT * alpha;

        // Wrap theta to -pi to pi
        if (theta > M_PI) theta -= 2 * M_PI;
        if (theta < -M_PI) theta += 2 * M_PI;

        // Write new state back to global memory
        state[idx*2] = theta;
        state[idx*2+1] = omega;
    }
    
    final_state[idx*2] = theta;
    final_state[idx*2+1] = omega;
}

int main() {
    double *d_state;
    hiprandState_t *d_states;
    double *d_initial_state;
    double *d_final_state;
    
    hipError_t error1, error2, error3, error4;
    error1 = hipMalloc((void**)&d_state, NUM_SIMULATIONS*2*sizeof(double));
    error2 = hipMalloc((void**)&d_states, NUM_SIMULATIONS*sizeof(hiprandState_t));
    error3 = hipMalloc((void**)&d_initial_state, NUM_SIMULATIONS*2*sizeof(double));
    error4 = hipMalloc((void**)&d_final_state, NUM_SIMULATIONS*2*sizeof(double));
    
    if (error1 != hipSuccess) {
        fprintf(stderr, "cudaMalloc1 failed: %s\n", hipGetErrorString(error1));
    }
    else if (error2 != hipSuccess) {
        fprintf(stderr, "cudaMalloc2 failed: %s\n", hipGetErrorString(error2));
    }
    else if (error3 != hipSuccess) {
        fprintf(stderr, "cudaMalloc3 failed: %s\n", hipGetErrorString(error3));
    }
    else if (error4 != hipSuccess) {
        fprintf(stderr, "cudaMalloc4 failed: %s\n", hipGetErrorString(error4));
    }


    init<<<(NUM_SIMULATIONS + 255) / 256, 256>>>(time(NULL), d_states);

    simulate <<< (NUM_SIMULATIONS + 255) / 256, 256 >>> (d_states, d_state, d_initial_state, d_final_state);

    double *h_initial_state = new double[NUM_SIMULATIONS*2];
    double *h_final_state = new double[NUM_SIMULATIONS*2];

    hipMemcpy(h_initial_state, d_initial_state, NUM_SIMULATIONS*2*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_final_state, d_final_state, NUM_SIMULATIONS*2*sizeof(double), hipMemcpyDeviceToHost);

    // Write initial states to binary file
    std::ofstream initial_file("initial_states.bin", std::ios::binary);
    initial_file.write(reinterpret_cast<char*>(h_initial_state), NUM_SIMULATIONS*2*sizeof(double));
    initial_file.close();

    // Write final states to binary file
    std::ofstream final_file("final_states.bin", std::ios::binary);
    final_file.write(reinterpret_cast<char*>(h_final_state), NUM_SIMULATIONS*2*sizeof(double));
    final_file.close();

    delete[] h_initial_state;
    delete[] h_final_state;
    hipFree(d_state);
    hipFree(d_states);
    hipFree(d_initial_state);
    hipFree(d_final_state);
    return 0;
}


